#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <cstring>
#include <cmath>
#include <ctime>
#include <pthread.h>
#include <omp.h>
#include <string>
#include <vector>
#include <algorithm>

extern "C" {
#include "../include/golomb.h"
}

/* ---------------- Checkpointing header ---------------- */
typedef struct {
    char     magic[4];   // "GRCP"
    uint32_t version;    // 1
    uint32_t n;
    uint32_t L;
    uint64_t total;
    uint32_t hint_s;
    uint32_t hint_t;
    uint32_t hint_used;  // 0/1
} cp_header_t;

static int cp_load_file(const char *path,
                        int n,
                        int target_length,
                        long long total,
                        int hint_s,
                        int hint_t,
                        int hint_used,
                        uint32_t *done_words,
                        size_t words)
{
    FILE *fp = fopen(path, "rb");
    if (!fp) return 0;
    cp_header_t h;
    size_t r = fread(&h, 1, sizeof h, fp);
    if (r != sizeof h || memcmp(h.magic, "GRCP", 4) != 0 || h.version != 1) { fclose(fp); return 0; }
    if (h.n != (uint32_t)n || h.L != (uint32_t)target_length || h.total != (uint64_t)total) { fclose(fp); return 0; }
    if (h.hint_s != (uint32_t)hint_s || h.hint_t != (uint32_t)hint_t || h.hint_used != (uint32_t)hint_used) { fclose(fp); return 0; }
    size_t want = words * sizeof(uint32_t);
    r = fread(done_words, 1, want, fp);
    fclose(fp);
    return r == want;
}

static int cp_save_file(const char *path,
                        int n,
                        int target_length,
                        long long total,
                        int hint_s,
                        int hint_t,
                        int hint_used,
                        const uint32_t *done_words,
                        size_t words)
{
    char tmp[1024];
    snprintf(tmp, sizeof tmp, "%s.tmp", path);
    FILE *fp = fopen(tmp, "wb");
    if (!fp) return 0;
    cp_header_t h;
    memcpy(h.magic, "GRCP", 4);
    h.version = 1;
    h.n = (uint32_t)n;
    h.L = (uint32_t)target_length;
    h.total = (uint64_t)total;
    h.hint_s = (uint32_t)hint_s;
    h.hint_t = (uint32_t)hint_t;
    h.hint_used = (uint32_t)hint_used;
    size_t w1 = fwrite(&h, 1, sizeof h, fp);
    size_t w2 = fwrite(done_words, 1, words * sizeof(uint32_t), fp);
    int ok = (w1 == sizeof h) && (w2 == words * sizeof(uint32_t));
    if (fclose(fp) != 0) ok = 0;
    if (!ok) { remove(tmp); return 0; }
    if (rename(tmp, path) != 0) { remove(tmp); return 0; }
    return 1;
}

/* ---------------- Scalar DFS (no SIMD) ---------------- */
static inline void set_bit64(uint64_t *bs, int idx) { bs[idx >> 6] |= 1ULL << (idx & 63); }
static inline void clr_bit64(uint64_t *bs, int idx) { bs[idx >> 6] &= ~(1ULL << (idx & 63)); }
static inline int  test_bit64(const uint64_t *bs, int idx) { return (bs[idx >> 6] >> (idx & 63)) & 1ULL; }

static bool dfs_scalar(int depth, int n, int target_len, int *pos, uint64_t *dist_bs, bool verbose)
{
    if (depth == n) return pos[n - 1] == target_len;
    int last = pos[depth - 1];
    if (last + (n - depth) > target_len) return false;
    int max_next = target_len - (n - depth - 1);
    if (depth == 1) {
        int limit = target_len / 2;
        if (limit < last + 1) limit = last + 1;
        if (max_next > limit) max_next = limit;
    }
    for (int next = last + 1; next <= max_next; ++next) {
        bool ok = true;
        for (int i = 0; i < depth; ++i) {
            int d = next - pos[i];
            if (test_bit64(dist_bs, d)) { ok = false; break; }
        }
        if (!ok) continue;
        pos[depth] = next;
        for (int i = 0; i < depth; ++i) set_bit64(dist_bs, next - pos[i]);
        if (verbose && depth < 6) { printf("depth %d add %d\n", depth, next); }
        if (dfs_scalar(depth + 1, n, target_len, pos, dist_bs, verbose)) return true;
        for (int i = 0; i < depth; ++i) clr_bit64(dist_bs, next - pos[i]);
    }
    return false;
}

/* ---------------- GPU candidate prefilter ---------------- */
struct Cand { int s, t, score; };

__global__ void prefilter_kernel(int n, int target_len, const Cand *cands, int64_t total, unsigned char *ok)
{
    int64_t i = blockIdx.x * 1LL * blockDim.x + threadIdx.x;
    if (i >= total) return;
    int s = cands[i].s;
    int t = cands[i].t;
    int max_next = target_len - (n - 3 - 1); // depth=3 => max_next = L - (n-4)
    if (max_next <= t) { ok[i] = 0; return; }
    // Existing distances: s, t, t-s
    int ds23 = t - s;
    unsigned char good = 0;
    for (int next = t + 1; next <= max_next; ++next) {
        int d0 = next;           // next - 0
        int d1 = next - s;       // next - s
        int d2 = next - t;       // next - t
        // check against existing distances
        if (d0 == s || d0 == t || d0 == ds23) continue;
        if (d1 == s || d1 == t || d1 == ds23) continue;
        if (d2 == s || d2 == t || d2 == ds23) continue;
        // also ensure d0, d1, d2 are pairwise distinct (they are, since 0 < s < t < next)
        good = 1; break;
    }
    ok[i] = good;
}

/* ---------------- Heartbeat ---------------- */
static volatile int g_done = 0;
static volatile int g_current_L = -1;
static double g_vt_sec = 0.0;
static struct timespec g_ts_start;

static void *heartbeat_thread(void *)
{
    while (!g_done) {
        struct timespec ts_now; clock_gettime(CLOCK_MONOTONIC, &ts_now);
        double since = (ts_now.tv_sec - g_ts_start.tv_sec) + (ts_now.tv_nsec - g_ts_start.tv_nsec) / 1e9;
        int L = g_current_L;
        if (g_vt_sec > 0.0 && L >= 0) {
            // format mm:ss.mmm
            int minutes = (int)(since / 60.0);
            double seconds = since - minutes * 60.0;
            if (minutes > 0) printf("[VT] %02d:%06.3f elapsed – current L=%d\n", minutes, seconds, L);
            else              printf("[VT] %.3f s elapsed – current L=%d\n", seconds, L);
            fflush(stdout);
        }
        struct timespec req = { (time_t)g_vt_sec, (long)((g_vt_sec - (time_t)g_vt_sec) * 1e9) };
        nanosleep(&req, NULL);
    }
    return NULL;
}

/* ---------------- Main (CUDA-enhanced mp) ---------------- */
int main(int argc, char **argv)
{
    if (argc < 2) {
        fprintf(stderr, "Usage: %s <n> [-b] [-v] [-f <file>] [-fi <sec>] [-vt <min>]\n", argv[0]);
        return 1;
    }
    int n = atoi(argv[1]);
    bool verbose = false;
    bool use_b = false;
    const char *cp_path = NULL;
    int cp_interval = 60;
    double vt_min = 0.0;

    for (int i = 2; i < argc; ++i) {
        if (strcmp(argv[i], "-v") == 0) verbose = true;
        else if (strcmp(argv[i], "-b") == 0) use_b = true;
        else if (strcmp(argv[i], "-f") == 0 && i + 1 < argc) { cp_path = argv[++i]; }
        else if (strcmp(argv[i], "-fi") == 0 && i + 1 < argc) { cp_interval = atoi(argv[++i]); if (cp_interval <= 0) cp_interval = 60; }
        else if (strcmp(argv[i], "-vt") == 0 && i + 1 < argc) { vt_min = atof(argv[++i]); }
        else {
            fprintf(stderr, "Unknown or incomplete option: %s\n", argv[i]);
            return 2;
        }
    }

    const ruler_t *ref = lut_lookup_by_marks(n);
    if (!use_b || !ref) {
        fprintf(stderr, "This CUDA variant currently requires -b and a known LUT length for n=%d.\n", n);
        return 3;
    }
    int target_length = ref->length;

    // Start time and heartbeat
    clock_gettime(CLOCK_MONOTONIC, &g_ts_start);
    time_t t_start_wall = time(NULL);
    char start_iso[32]; strftime(start_iso, sizeof start_iso, "%F %T", localtime(&t_start_wall));
    printf("Start time: %s\n", start_iso);
    g_current_L = target_length;
    g_vt_sec = vt_min > 0 ? vt_min * 60.0 : 0.0;
    pthread_t hb_th; if (g_vt_sec > 0.0) pthread_create(&hb_th, NULL, heartbeat_thread, NULL);

    // Build candidates like -mp
    int half = target_length / 2;
    int T = target_length - (n - 2);
    int second_max = half; if (second_max > T - 1) second_max = T - 1; if (second_max < 1) second_max = 1;

    long long total = 0;
    for (int s = 1; s <= second_max; ++s) {
        int cnt = T - s; if (cnt > 0) total += cnt;
    }
    std::vector<Cand> cands; cands.reserve((size_t)total);
    int use_hint_order = (ref && getenv("GOLOMB_NO_HINTS") == NULL) ? 1 : 0;
    for (int s = 1; s <= second_max; ++s) {
        for (int t = s + 1; t <= T; ++t) {
            int score = 0;
            if (use_hint_order) {
                int ds = s - ref->pos[1]; if (ds < 0) ds = -ds;
                int dt = t - ref->pos[2]; if (dt < 0) dt = -dt;
                score = ds + dt;
            }
            cands.push_back({s, t, score});
        }
    }
    if (use_hint_order && cands.size() > 1) {
        std::stable_sort(cands.begin(), cands.end(), [](const Cand &a, const Cand &b){
            if (a.score != b.score) return a.score < b.score;
            if (a.s != b.s) return a.s < b.s;
            return a.t < b.t;
        });
    }

    // Checkpoint bitset
    size_t words = (size_t)((total + 31) / 32); if (words == 0) words = 1;
    std::vector<uint32_t> done_words(words, 0);
    if (cp_path && *cp_path) {
        int hs = use_hint_order && ref ? ref->pos[1] : 0;
        int ht = use_hint_order && ref ? ref->pos[2] : 0;
        (void)cp_load_file(cp_path, n, target_length, total, hs, ht, use_hint_order, done_words.data(), words);
        (void)cp_save_file(cp_path, n, target_length, total, hs, ht, use_hint_order, done_words.data(), words);
    }

    // Optional GPU prefilter: mark candidates that can advance one more mark without immediate duplicates
    int device_count = 0;
    hipError_t derr = hipGetDeviceCount(&device_count);
    int rt_ver = 0, dr_ver = 0; hipRuntimeGetVersion(&rt_ver); hipDriverGetVersion(&dr_ver);
    const char *cvd = getenv("CUDA_VISIBLE_DEVICES");
    printf("[CUDA] Runtime=%d Driver=%d CUDA_VISIBLE_DEVICES=%s\n", rt_ver, dr_ver, cvd ? cvd : "(unset)");
    if (derr != hipSuccess) {
        printf("[CUDA] hipGetDeviceCount error: %s (%d)\n", hipGetErrorString(derr), (int)derr);
    }
    if (device_count > 0) {
        int dev = 0;
        hipError_t sderr = hipSetDevice(dev);
        if (sderr != hipSuccess) {
            printf("[CUDA] hipSetDevice(%d) failed: %s (%d)\n", dev, hipGetErrorString(sderr), (int)sderr);
        }
        // create context
        hipFree(0);
        hipDeviceProp_t prop; hipGetDeviceProperties(&prop, dev);
        printf("[CUDA] Using device %d: %s\n", dev, prop.name);
    } else {
        printf("[CUDA] No CUDA device found – running CPU-only prefilter.\n");
    }
    if (device_count > 0 && total > 0) {
        Cand *d_cands = nullptr; unsigned char *d_ok = nullptr; unsigned char *h_ok = (unsigned char*)malloc((size_t)total);
        hipMalloc(&d_cands, sizeof(Cand) * (size_t)total);
        hipMalloc(&d_ok, (size_t)total);
        hipMemcpy(d_cands, cands.data(), sizeof(Cand) * (size_t)total, hipMemcpyHostToDevice);
        int threads = 256; int blocks = (int)((total + threads - 1) / threads);
        prefilter_kernel<<<blocks, threads>>>(n, target_length, d_cands, total, d_ok);
        hipDeviceSynchronize();
        hipMemcpy(h_ok, d_ok, (size_t)total, hipMemcpyDeviceToHost);
        // Rebuild candidate order: keep original order, but place ok==1 first
        {
            std::vector<Cand> reordered; reordered.reserve((size_t)total);
            size_t ok_cnt = 0;
            // first pass: ok == 1
            for (size_t i = 0; i < (size_t)total; ++i) {
                if (h_ok[i]) { reordered.push_back(cands[i]); ++ok_cnt; }
            }
            // second pass: ok == 0
            for (size_t i = 0; i < (size_t)total; ++i) {
                if (!h_ok[i]) reordered.push_back(cands[i]);
            }
            cands.swap(reordered);
            printf("[CUDA] Prefiltered %lld candidates: %zu pass first check.\n", total, ok_cnt);
        }
        hipFree(d_cands); hipFree(d_ok); free(h_ok);
    }

    // Fast-lane: try exact LUT pair on CPU first if hints enabled
    volatile int found = 0; ruler_t res_local{};
    if (ref && getenv("GOLOMB_NO_HINTS") == NULL) {
        int s0 = ref->pos[1], t0 = ref->pos[2];
        if (s0 >= 1 && s0 <= second_max && t0 > s0 && t0 <= T) {
            uint64_t bs[BS_WORDS] = {0};
            int pos[MAX_MARKS]; pos[0] = 0; pos[1] = s0; pos[2] = t0;
            set_bit64(bs, s0);
            int d13 = t0; int d23 = t0 - s0;
            if (!test_bit64(bs, d13) && !test_bit64(bs, d23)) {
                set_bit64(bs, d13); set_bit64(bs, d23);
                if (dfs_scalar(3, n, target_length, pos, bs, verbose)) {
                    res_local.marks = n; res_local.length = pos[n - 1]; memcpy(res_local.pos, pos, n * sizeof(int)); found = 1;
                }
            }
        }
    }

    struct timespec ts_last_flush; clock_gettime(CLOCK_MONOTONIC, &ts_last_flush);

    // Parallel CPU search across candidates (static mp), respecting checkpoint
    #pragma omp parallel for schedule(dynamic, 16)
    for (long long i = 0; i < total; ++i) {
        if (found) continue;
        // skip processed
        if (cp_path && *cp_path) {
            size_t wi = (size_t)(i >> 5); uint32_t mask = 1u << (i & 31);
            if (done_words[wi] & mask) continue;
        }
        int second = cands[(size_t)i].s;
        int third  = cands[(size_t)i].t;
        uint64_t bs[BS_WORDS] = {0};
        int pos[MAX_MARKS]; pos[0] = 0; pos[1] = second; pos[2] = third;
        set_bit64(bs, second);
        int d13 = third; int d23 = third - second;
        if (test_bit64(bs, d13) || test_bit64(bs, d23)) goto mark_done;
        set_bit64(bs, d13); set_bit64(bs, d23);
        if (dfs_scalar(3, n, target_length, pos, bs, verbose)) {
            #pragma omp critical
            {
                if (!found) { res_local.marks = n; res_local.length = pos[n - 1]; memcpy(res_local.pos, pos, n * sizeof(int)); found = 1; }
            }
        }
    mark_done:
        if (cp_path && *cp_path) {
            size_t wi = (size_t)(i >> 5); uint32_t mask = 1u << (i & 31);
            __sync_fetch_and_or(&done_words[wi], mask);
            struct timespec ts_now; clock_gettime(CLOCK_MONOTONIC, &ts_now);
            time_t dt = ts_now.tv_sec - ts_last_flush.tv_sec;
            if (dt >= cp_interval) {
                #pragma omp critical
                {
                    struct timespec ts_chk; clock_gettime(CLOCK_MONOTONIC, &ts_chk);
                    if (ts_chk.tv_sec - ts_last_flush.tv_sec >= cp_interval) {
                        int hs2 = use_hint_order && ref ? ref->pos[1] : 0;
                        int ht2 = use_hint_order && ref ? ref->pos[2] : 0;
                        (void)cp_save_file(cp_path, n, target_length, total, hs2, ht2, use_hint_order, done_words.data(), words);
                        ts_last_flush = ts_chk;
                    }
                }
            }
        }
    }

    if (cp_path && *cp_path) {
        int hs = use_hint_order && ref ? ref->pos[1] : 0;
        int ht = use_hint_order && ref ? ref->pos[2] : 0;
        (void)cp_save_file(cp_path, n, target_length, total, hs, ht, use_hint_order, done_words.data(), words);
    }

    g_done = 1; if (g_vt_sec > 0.0) pthread_join(hb_th, NULL);

    if (found) {
        // print ruler
        print_ruler(&res_local);
        return 0;
    }
    fprintf(stderr, "No ruler found at L=%d (unexpected for LUT-verified -b).\n", target_length);
    return 1;
}
