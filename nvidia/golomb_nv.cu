#include "hip/hip_runtime.h"
#include "glibc_c23_math_compat.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <cstring>
#include <ctime>
#include <pthread.h>
#include <omp.h>
#include <string>
#include <vector>
#include <algorithm>

extern "C" {
#include "../include/golomb.h"
}

// Optional hint from LUT to guide DFS next-choice ordering
static const ruler_t *g_ref_hint = NULL;

/* ---------------- Checkpointing header ---------------- */
typedef struct {
    char     magic[4];   // "GRCP"
    uint32_t version;    // 1
    uint32_t n;
    uint32_t L;
    uint64_t total;
    uint32_t hint_s;
    uint32_t hint_t;
    uint32_t hint_used;  // 0/1
} cp_header_t;

static int cp_load_file(const char *path,
                        int n,
                        int target_length,
                        long long total,
                        int hint_s,
                        int hint_t,
                        int hint_used,
                        uint32_t *done_words,
                        size_t words)
{
    FILE *fp = fopen(path, "rb");
    if (!fp) return 0;
    cp_header_t h;
    size_t r = fread(&h, 1, sizeof h, fp);
    if (r != sizeof h || memcmp(h.magic, "GRCP", 4) != 0 || h.version != 1) { fclose(fp); return 0; }
    if (h.n != (uint32_t)n || h.L != (uint32_t)target_length || h.total != (uint64_t)total) { fclose(fp); return 0; }
    if (h.hint_s != (uint32_t)hint_s || h.hint_t != (uint32_t)hint_t || h.hint_used != (uint32_t)hint_used) { fclose(fp); return 0; }
    size_t want = words * sizeof(uint32_t);
    r = fread(done_words, 1, want, fp);
    fclose(fp);
    return r == want;
}

static int cp_save_file(const char *path,
                        int n,
                        int target_length,
                        long long total,
                        int hint_s,
                        int hint_t,
                        int hint_used,
                        const uint32_t *done_words,
                        size_t words)
{
    char tmp[1024];
    snprintf(tmp, sizeof tmp, "%s.tmp", path);
    FILE *fp = fopen(tmp, "wb");
    if (!fp) return 0;
    cp_header_t h;
    memcpy(h.magic, "GRCP", 4);
    h.version = 1;
    h.n = (uint32_t)n;
    h.L = (uint32_t)target_length;
    h.total = (uint64_t)total;
    h.hint_s = (uint32_t)hint_s;
    h.hint_t = (uint32_t)hint_t;
    h.hint_used = (uint32_t)hint_used;
    size_t w1 = fwrite(&h, 1, sizeof h, fp);
    size_t w2 = fwrite(done_words, 1, words * sizeof(uint32_t), fp);
    int ok = (w1 == sizeof h) && (w2 == words * sizeof(uint32_t));
    if (fclose(fp) != 0) ok = 0;
    if (!ok) { remove(tmp); return 0; }
    if (rename(tmp, path) != 0) { remove(tmp); return 0; }
    return 1;
}

/* ---------------- Scalar DFS (no SIMD) ---------------- */
static inline void set_bit64(uint64_t *bs, int idx) { bs[idx >> 6] |= 1ULL << (idx & 63); }
static inline void clr_bit64(uint64_t *bs, int idx) { bs[idx >> 6] &= ~(1ULL << (idx & 63)); }
static inline int  test_bit64(const uint64_t *bs, int idx) { return (bs[idx >> 6] >> (idx & 63)) & 1ULL; }

static bool dfs_scalar(int depth, int n, int target_len, int *pos, uint64_t *dist_bs, bool verbose)
{
    if (depth == n) return pos[n - 1] == target_len;
    int last = pos[depth - 1];
    // Stronger lower-bound prune: to place the remaining (n - depth) marks,
    // we need at least a triangular number of additional length.
    int rem = n - depth; // marks remaining to place, including the final one
    int tri_needed = (rem * (rem - 1)) / 2; // 1 + 2 + ... + (rem-1)
    if (last + tri_needed > target_len) return false;
    // Tighten the max_next bound to preserve headroom for the remaining marks after choosing 'next'
    int rem_after = rem - 1;
    int tri_after = (rem_after * (rem_after - 1)) / 2; // minimal extra beyond 'next'
    int max_next = target_len - tri_after;
    if (depth == 1) {
        int limit = target_len / 2;
        if (limit < last + 1) limit = last + 1;
        if (max_next > limit) max_next = limit;
    }
    // Try LUT-hinted next position first, if available and valid
    int hint_val = -1;
    if (g_ref_hint && depth < g_ref_hint->marks) {
        hint_val = g_ref_hint->pos[depth];
    }
    if (hint_val > last && hint_val <= max_next) {
        bool ok = true;
        for (int i = 0; i < depth; ++i) {
            int d = hint_val - pos[i];
            if (test_bit64(dist_bs, d)) { ok = false; break; }
        }
        if (ok) {
            pos[depth] = hint_val;
            for (int i = 0; i < depth; ++i) set_bit64(dist_bs, hint_val - pos[i]);
            if (verbose && depth < 6) { fprintf(stderr, "depth %d add %d (hint)\n", depth, hint_val); }
            if (dfs_scalar(depth + 1, n, target_len, pos, dist_bs, verbose)) return true;
            for (int i = 0; i < depth; ++i) clr_bit64(dist_bs, hint_val - pos[i]);
        }
    }
    for (int next = last + 1; next <= max_next; ++next) {
        if (next == hint_val) continue; // already tried
        bool ok = true;
        for (int i = 0; i < depth; ++i) {
            int d = next - pos[i];
            if (test_bit64(dist_bs, d)) { ok = false; break; }
        }
        if (!ok) continue;
        pos[depth] = next;
        for (int i = 0; i < depth; ++i) set_bit64(dist_bs, next - pos[i]);
        if (verbose && depth < 6) { fprintf(stderr, "depth %d add %d\n", depth, next); }
        if (dfs_scalar(depth + 1, n, target_len, pos, dist_bs, verbose)) return true;
        for (int i = 0; i < depth; ++i) clr_bit64(dist_bs, next - pos[i]);
    }
    return false;
}

/* ---------------- GPU candidate prefilter ---------------- */
struct Cand { int s, t, score; };

__global__ void prefilter_kernel(int n, int L, const Cand *cands, int64_t total, unsigned char *ok)
{
    int64_t i = blockIdx.x * 1LL * blockDim.x + threadIdx.x;
    if (i >= total) return;
    int s = cands[i].s;
    int t = cands[i].t;
    // Depth=3 state: pos[0]=0, pos[1]=s, pos[2]=t
    int rem = n - 3; // remaining marks including final
    int tri_needed = rem * (rem - 1) / 2; // minimal additional length needed after 't'
    if (t + tri_needed > L) { ok[i] = 0; return; }

    // existing distances at depth=3
    int d_s = s;
    int d_t = t;
    int d_st = t - s;

    // First next bound using triangular after-placing bound
    int rem_after1 = rem - 1;                    // after choosing u
    int tri_after1 = rem_after1 * (rem_after1 - 1) / 2;
    int max_u = L - tri_after1;
    if (max_u <= t) { ok[i] = 0; return; }

    unsigned char ok1 = 0, ok2 = 0;
    for (int u = t + 1; u <= max_u; ++u) {
        int du0 = u;       // u - 0
        int du1 = u - s;   // u - s
        int du2 = u - t;   // u - t
        // uniqueness vs existing
        if (du0 == d_s || du0 == d_t || du0 == d_st) continue;
        if (du1 == d_s || du1 == d_t || du1 == d_st) continue;
        if (du2 == d_s || du2 == d_t || du2 == d_st) continue;
        // pairwise distinct among du0,du1,du2 always true as 0<s<t<u
        ok1 = 1; // one-step feasible

        // Two-step feasibility: try to place v > u
        int rem2 = rem_after1 - 1;                 // remaining after placing u
        int tri_needed2 = rem2 * (rem2 - 1) / 2;   // minimal addl length after v must be <= L - v
        if (u + tri_needed2 > L) continue;
        int rem_after2 = rem2 - 1;
        int tri_after2 = rem_after2 * (rem_after2 - 1) / 2;
        int max_v = L - tri_after2;
        if (max_v <= u) continue;

        // Distances present after u:
        // {d_s, d_t, d_st, du0, du1, du2}
        for (int v = u + 1; v <= max_v; ++v) {
            int dv0 = v;       // v - 0
            int dv1 = v - s;   // v - s
            int dv2 = v - t;   // v - t
            int dv3 = v - u;   // v - u
            // compare to existing set
            if (dv0 == d_s || dv0 == d_t || dv0 == d_st || dv0 == du0 || dv0 == du1 || dv0 == du2) continue;
            if (dv1 == d_s || dv1 == d_t || dv1 == d_st || dv1 == du0 || dv1 == du1 || dv1 == du2) continue;
            if (dv2 == d_s || dv2 == d_t || dv2 == d_st || dv2 == du0 || dv2 == du1 || dv2 == du2) continue;
            if (dv3 == d_s || dv3 == d_t || dv3 == d_st || dv3 == du0 || dv3 == du1 || dv3 == du2) continue;
            // pairwise distinct among dv0..dv3; trivial order ensures dv0>dv1>dv2>dv3>0 but check collisions among dv* themselves:
            if (dv0 == dv1 || dv0 == dv2 || dv0 == dv3) continue;
            if (dv1 == dv2 || dv1 == dv3) continue;
            if (dv2 == dv3) continue;
            ok2 = 2; break;
        }
        if (ok2) break;
    }
    ok[i] = ok2 ? ok2 : ok1;
}

/* ---------------- Heartbeat ---------------- */
static volatile int g_done = 0;
static volatile int g_current_L = -1;
static double g_vt_sec = 0.0;
static struct timespec g_ts_start;

static void *heartbeat_thread(void *)
{
    while (!g_done) {
        struct timespec ts_now; clock_gettime(CLOCK_MONOTONIC, &ts_now);
        double since = (ts_now.tv_sec - g_ts_start.tv_sec) + (ts_now.tv_nsec - g_ts_start.tv_nsec) / 1e9;
        int L = g_current_L;
        if (g_vt_sec > 0.0 && L >= 0) {
            // format mm:ss.mmm
            int minutes = (int)(since / 60.0);
            double seconds = since - minutes * 60.0;
            if (minutes > 0) fprintf(stderr, "[VT] %02d:%06.3f elapsed – current L=%d\n", minutes, seconds, L);
            else              fprintf(stderr, "[VT] %.3f s elapsed – current L=%d\n", seconds, L);
            fflush(stderr);
        }
        struct timespec req = { (time_t)g_vt_sec, (long)((g_vt_sec - (time_t)g_vt_sec) * 1e9) };
        nanosleep(&req, NULL);
    }
    return NULL;
}

/* ---------------- Main (CUDA-enhanced mp) ---------------- */
int main(int argc, char **argv)
{
    if (argc < 2) {
        fprintf(stderr, "Usage: %s <n> [-b] [-v] [-H] [-f <file>] [-fi <sec>] [-vt <min>]\n", argv[0]);
        return 1;
    }
    int n = atoi(argv[1]);
    bool verbose = false;
    bool use_b = false;
    bool hints = false; // enable LUT hint order and fast-lane only with -H
    const char *cp_path = NULL;
    int cp_interval = 60;
    double vt_min = 0.0;

    for (int i = 2; i < argc; ++i) {
        if (strcmp(argv[i], "-v") == 0) verbose = true;
        else if (strcmp(argv[i], "-b") == 0) use_b = true;
        else if (strcmp(argv[i], "-H") == 0) hints = true;
        else if (strcmp(argv[i], "-f") == 0 && i + 1 < argc) { cp_path = argv[++i]; }
        else if (strcmp(argv[i], "-fi") == 0 && i + 1 < argc) { cp_interval = atoi(argv[++i]); if (cp_interval <= 0) cp_interval = 60; }
        else if (strcmp(argv[i], "-vt") == 0 && i + 1 < argc) { vt_min = atof(argv[++i]); }
        else {
            fprintf(stderr, "Unknown or incomplete option: %s\n", argv[i]);
            return 2;
        }
    }

    const ruler_t *ref = lut_lookup_by_marks(n);
    if (!use_b || !ref) {
        fprintf(stderr, "This CUDA variant currently requires -b and a known LUT length for n=%d.\n", n);
        return 3;
    }
    int target_length = ref->length;

    // Start time and heartbeat
    clock_gettime(CLOCK_MONOTONIC, &g_ts_start);
    time_t t_start_wall = time(NULL);
    char start_iso[32]; strftime(start_iso, sizeof start_iso, "%F %T", localtime(&t_start_wall));
    fprintf(stderr, "Start time: %s\n", start_iso);
    g_current_L = target_length;
    g_vt_sec = vt_min > 0 ? vt_min * 60.0 : 0.0;
    pthread_t hb_th; if (g_vt_sec > 0.0) pthread_create(&hb_th, NULL, heartbeat_thread, NULL);

    // Build candidates like -mp
    int half = target_length / 2;
    int T = target_length - (n - 2);
    int second_max = half; if (second_max > T - 1) second_max = T - 1; if (second_max < 1) second_max = 1;

    long long total = 0;
    for (int s = 1; s <= second_max; ++s) {
        int cnt = T - s; if (cnt > 0) total += cnt;
    }
    std::vector<Cand> cands; cands.reserve((size_t)total);
    int use_hint_order = (hints && ref && getenv("GOLOMB_NO_HINTS") == NULL) ? 1 : 0;
    g_ref_hint = use_hint_order ? ref : NULL;
    for (int s = 1; s <= second_max; ++s) {
        for (int t = s + 1; t <= T; ++t) {
            int score = 0;
            if (use_hint_order) {
                int ds = s - ref->pos[1]; if (ds < 0) ds = -ds;
                int dt = t - ref->pos[2]; if (dt < 0) dt = -dt;
                score = ds + dt;
            }
            cands.push_back({s, t, score});
        }
    }
    if (use_hint_order && cands.size() > 1) {
        std::stable_sort(cands.begin(), cands.end(), [](const Cand &a, const Cand &b){
            if (a.score != b.score) return a.score < b.score;
            if (a.s != b.s) return a.s < b.s;
            return a.t < b.t;
        });
    }

    // Checkpoint bitset
    size_t words = (size_t)((total + 31) / 32); if (words == 0) words = 1;
    std::vector<uint32_t> done_words(words, 0);
    if (cp_path && *cp_path) {
        int hs = use_hint_order && ref ? ref->pos[1] : 0;
        int ht = use_hint_order && ref ? ref->pos[2] : 0;
        (void)cp_load_file(cp_path, n, target_length, total, hs, ht, use_hint_order, done_words.data(), words);
        (void)cp_save_file(cp_path, n, target_length, total, hs, ht, use_hint_order, done_words.data(), words);
    }

    // Optional GPU prefilter: mark candidates that can advance one more mark without immediate duplicates
    int device_count = 0;
    hipError_t derr = hipGetDeviceCount(&device_count);
    int rt_ver = 0, dr_ver = 0; hipRuntimeGetVersion(&rt_ver); hipDriverGetVersion(&dr_ver);
    const char *cvd = getenv("CUDA_VISIBLE_DEVICES");
    fprintf(stderr, "[CUDA] Runtime=%d Driver=%d CUDA_VISIBLE_DEVICES=%s\n", rt_ver, dr_ver, cvd ? cvd : "(unset)");
    if (derr != hipSuccess) {
        fprintf(stderr, "[CUDA] hipGetDeviceCount error: %s (%d)\n", hipGetErrorString(derr), (int)derr);
    }
    if (device_count > 0) {
        int dev = 0;
        hipError_t sderr = hipSetDevice(dev);
        if (sderr != hipSuccess) {
            fprintf(stderr, "[CUDA] hipSetDevice(%d) failed: %s (%d)\n", dev, hipGetErrorString(sderr), (int)sderr);
        }
        // create context
        hipFree(0);
        hipDeviceProp_t prop; hipGetDeviceProperties(&prop, dev);
        fprintf(stderr, "[CUDA] Using device %d: %s\n", dev, prop.name);
    } else {
        fprintf(stderr, "[CUDA] No CUDA device found – running CPU-only prefilter.\n");
    }
    if (device_count > 0 && total > 0) {
        Cand *d_cands = nullptr; unsigned char *d_ok = nullptr; unsigned char *h_ok = (unsigned char*)malloc((size_t)total);
        hipMalloc(&d_cands, sizeof(Cand) * (size_t)total);
        hipMalloc(&d_ok, (size_t)total);
        hipMemcpy(d_cands, cands.data(), sizeof(Cand) * (size_t)total, hipMemcpyHostToDevice);
        int threads = 256; int blocks = (int)((total + threads - 1) / threads);
        prefilter_kernel<<<blocks, threads>>>(n, target_length, d_cands, total, d_ok);
        hipError_t kerr = hipGetLastError();
        if (kerr != hipSuccess) {
            fprintf(stderr, "[CUDA] prefilter kernel launch error: %s (%d)\n", hipGetErrorString(kerr), (int)kerr);
        }
        hipDeviceSynchronize();
        hipMemcpy(h_ok, d_ok, (size_t)total, hipMemcpyDeviceToHost);
        // Rebuild candidate order: 2-step first, then 1-step, then 0
        {
            std::vector<Cand> reordered; reordered.reserve((size_t)total);
            size_t ok2_cnt = 0, ok1_cnt = 0;
            for (size_t i = 0; i < (size_t)total; ++i) {
                if (h_ok[i] >= 2) { reordered.push_back(cands[i]); ++ok2_cnt; }
            }
            for (size_t i = 0; i < (size_t)total; ++i) {
                if (h_ok[i] == 1) { reordered.push_back(cands[i]); ++ok1_cnt; }
            }
            for (size_t i = 0; i < (size_t)total; ++i) {
                if (h_ok[i] == 0) reordered.push_back(cands[i]);
            }
            cands.swap(reordered);
            fprintf(stderr, "[CUDA] Prefiltered %lld candidates: %zu two-step, %zu one-step.\n", total, ok2_cnt, ok1_cnt);
        }
        hipFree(d_cands); hipFree(d_ok); free(h_ok);
    }

    // Fast-lane: try exact LUT pair on CPU first only if hints are enabled (-H)
    volatile int found = 0; ruler_t res_local{};
    if (use_hint_order) {
        int s0 = ref->pos[1], t0 = ref->pos[2];
        if (s0 >= 1 && s0 <= second_max && t0 > s0 && t0 <= T) {
            uint64_t bs[BS_WORDS] = {0};
            int pos[MAX_MARKS]; pos[0] = 0; pos[1] = s0; pos[2] = t0;
            set_bit64(bs, s0);
            int d13 = t0; int d23 = t0 - s0;
            if (!test_bit64(bs, d13) && !test_bit64(bs, d23)) {
                set_bit64(bs, d13); set_bit64(bs, d23);
                if (dfs_scalar(3, n, target_length, pos, bs, verbose)) {
                    res_local.marks = n; res_local.length = pos[n - 1]; memcpy(res_local.pos, pos, n * sizeof(int)); found = 1;
                }
            }
        }
    }

    struct timespec ts_last_flush; clock_gettime(CLOCK_MONOTONIC, &ts_last_flush);

    // Parallel CPU search across candidates (static mp), respecting checkpoint
    #pragma omp parallel for schedule(guided, 1)
    for (long long i = 0; i < total; ++i) {
        if (found) continue;
        // skip processed
        if (cp_path && *cp_path) {
            size_t wi = (size_t)(i >> 5); uint32_t mask = 1u << (i & 31);
            if (done_words[wi] & mask) continue;
        }
        int second = cands[(size_t)i].s;
        int third  = cands[(size_t)i].t;
        uint64_t bs[BS_WORDS] = {0};
        int pos[MAX_MARKS]; pos[0] = 0; pos[1] = second; pos[2] = third;
        set_bit64(bs, second);
        int d13 = third; int d23 = third - second;
        if (test_bit64(bs, d13) || test_bit64(bs, d23)) goto mark_done;
        set_bit64(bs, d13); set_bit64(bs, d23);
        if (dfs_scalar(3, n, target_length, pos, bs, verbose)) {
            #pragma omp critical
            {
                if (!found) { res_local.marks = n; res_local.length = pos[n - 1]; memcpy(res_local.pos, pos, n * sizeof(int)); found = 1; }
            }
        }
    mark_done:
        if (cp_path && *cp_path) {
            size_t wi = (size_t)(i >> 5); uint32_t mask = 1u << (i & 31);
            __sync_fetch_and_or(&done_words[wi], mask);
            struct timespec ts_now; clock_gettime(CLOCK_MONOTONIC, &ts_now);
            time_t dt = ts_now.tv_sec - ts_last_flush.tv_sec;
            if (dt >= cp_interval) {
                #pragma omp critical
                {
                    struct timespec ts_chk; clock_gettime(CLOCK_MONOTONIC, &ts_chk);
                    if (ts_chk.tv_sec - ts_last_flush.tv_sec >= cp_interval) {
                        int hs2 = use_hint_order && ref ? ref->pos[1] : 0;
                        int ht2 = use_hint_order && ref ? ref->pos[2] : 0;
                        (void)cp_save_file(cp_path, n, target_length, total, hs2, ht2, use_hint_order, done_words.data(), words);
                        ts_last_flush = ts_chk;
                    }
                }
            }
        }
    }

    if (cp_path && *cp_path) {
        int hs = use_hint_order && ref ? ref->pos[1] : 0;
        int ht = use_hint_order && ref ? ref->pos[2] : 0;
        (void)cp_save_file(cp_path, n, target_length, total, hs, ht, use_hint_order, done_words.data(), words);
    }

    g_done = 1; if (g_vt_sec > 0.0) pthread_join(hb_th, NULL);

    if (found) {
        // Print in the same format as the C variant:
        // length=..\nmarks=..\npositions=..\ndistances=..\nmissing=..
        int L = res_local.length;
        int m = res_local.marks;
        printf("length=%d\nmarks=%d\npositions=", L, m);
        for (int i = 0; i < m; ++i) {
            printf("%d%s", res_local.pos[i], (i == m - 1) ? "" : " ");
        }
        // Build distance presence 1..L
        std::vector<unsigned char> present((size_t)L + 1, 0);
        for (int j = 0; j < m; ++j) {
            for (int i = 0; i < j; ++i) {
                int d = res_local.pos[j] - res_local.pos[i];
                if (d >= 1 && d <= L) present[(size_t)d] = 1;
            }
        }
        // distances line
        printf("\ndistances=");
        for (int d = 1, first = 1; d <= L; ++d) {
            if (present[(size_t)d]) {
                if (!first) putchar(' ');
                printf("%d", d);
                first = 0;
            }
        }
        // missing line
        printf("\nmissing=");
        for (int d = 1, first = 1; d <= L; ++d) {
            if (!present[(size_t)d]) {
                if (!first) putchar(' ');
                printf("%d", d);
                first = 0;
            }
        }
        putchar('\n');
        return 0;
    }
    fprintf(stderr, "No ruler found at L=%d (unexpected for LUT-verified -b).\n", target_length);
    return 1;
}
