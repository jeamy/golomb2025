#include "hip/hip_runtime.h"
#include "glibc_c23_math_compat.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <cstring>
#include <ctime>
#include <pthread.h>
#include <omp.h>
#include <string>
#include <vector>
#include <algorithm>

extern "C" {
#include "../include/golomb.h"
}

// (hint pointer removed; not needed)

/* ---------------- Checkpointing header ---------------- */
typedef struct {
    char     magic[4];   // "GRCP"
    uint32_t version;    // 1
    uint32_t n;
    uint32_t L;
    uint64_t total;
    uint32_t hint_s;
    uint32_t hint_t;
    uint32_t hint_used;  // 0/1
} cp_header_t;

static int cp_load_file(const char *path,
                        int n,
                        int target_length,
                        long long total,
                        int hint_s,
                        int hint_t,
                        int hint_used,
                        uint32_t *done_words,
                        size_t words)
{
    FILE *fp = fopen(path, "rb");
    if (!fp) return 0;
    cp_header_t h;
    size_t r = fread(&h, 1, sizeof h, fp);
    if (r != sizeof h || memcmp(h.magic, "GRCP", 4) != 0 || h.version != 1) { fclose(fp); return 0; }
    if (h.n != (uint32_t)n || h.L != (uint32_t)target_length || h.total != (uint64_t)total) { fclose(fp); return 0; }
    if (h.hint_s != (uint32_t)hint_s || h.hint_t != (uint32_t)hint_t || h.hint_used != (uint32_t)hint_used) { fclose(fp); return 0; }
    size_t want = words * sizeof(uint32_t);
    r = fread(done_words, 1, want, fp);
    fclose(fp);
    return r == want;
}

static int cp_save_file(const char *path,
                        int n,
                        int target_length,
                        long long total,
                        int hint_s,
                        int hint_t,
                        int hint_used,
                        const uint32_t *done_words,
                        size_t words)
{
    char tmp[1024];
    snprintf(tmp, sizeof tmp, "%s.tmp", path);
    FILE *fp = fopen(tmp, "wb");
    if (!fp) return 0;
    cp_header_t h;
    memcpy(h.magic, "GRCP", 4);
    h.version = 1;
    h.n = (uint32_t)n;
    h.L = (uint32_t)target_length;
    h.total = (uint64_t)total;
    h.hint_s = (uint32_t)hint_s;
    h.hint_t = (uint32_t)hint_t;
    h.hint_used = (uint32_t)hint_used;
    size_t w1 = fwrite(&h, 1, sizeof h, fp);
    size_t w2 = fwrite(done_words, 1, words * sizeof(uint32_t), fp);
    int ok = (w1 == sizeof h) && (w2 == words * sizeof(uint32_t));
    if (fclose(fp) != 0) ok = 0;
    if (!ok) { remove(tmp); return 0; }
    if (rename(tmp, path) != 0) { remove(tmp); return 0; }
    return 1;
}

/* ---------------- Host bitset helpers (match C solver) ---------------- */
static inline void set_bit64(uint64_t *bs, int idx) { bs[idx >> 6] |= 1ULL << (idx & 63); }
static inline int  test_bit64(const uint64_t *bs, int idx) { return (bs[idx >> 6] >> (idx & 63)) & 1ULL; }

/* ---------------- GPU candidate prefilter ---------------- */
struct Cand { int s, t, u_hint, score; };

__global__ void prefilter_kernel(int n, int L, Cand *cands, int64_t total, unsigned char *ok)
{
    int64_t i = blockIdx.x * 1LL * blockDim.x + threadIdx.x;
    if (i >= total) return;
    int s = cands[i].s;
    int t = cands[i].t;
    int best_u = 0;
    // Depth=3 state: pos[0]=0, pos[1]=s, pos[2]=t
    int rem = n - 3; // remaining marks including final
    int tri_needed = rem * (rem - 1) / 2; // minimal additional length needed after 't'
    if (t + tri_needed > L) { ok[i] = 0; return; }

    // existing distances at depth=3
    int d_s = s;
    int d_t = t;
    int d_st = t - s;

    // First next bound using triangular after-placing bound
    int rem_after1 = rem - 1;                    // after choosing u
    int tri_after1 = rem_after1 * (rem_after1 - 1) / 2;
    int max_u = L - tri_after1;
    if (max_u <= t) { ok[i] = 0; return; }

    unsigned char ok1 = 0, ok2 = 0;
    for (int u = t + 1; u <= max_u; ++u) {
        int du0 = u;       // u - 0
        int du1 = u - s;   // u - s
        int du2 = u - t;   // u - t
        // uniqueness vs existing
        if (du0 == d_s || du0 == d_t || du0 == d_st) continue;
        if (du1 == d_s || du1 == d_t || du1 == d_st) continue;
        if (du2 == d_s || du2 == d_t || du2 == d_st) continue;
        // pairwise distinct among du0,du1,du2 always true as 0<s<t<u
        ok1 = 1; // one-step feasible

        // Two-step feasibility: try to place v > u
        int rem2 = rem_after1 - 1;                 // remaining after placing u
        int tri_needed2 = rem2 * (rem2 - 1) / 2;   // minimal addl length after v must be <= L - v
        if (u + tri_needed2 > L) continue;
        int rem_after2 = rem2 - 1;
        int tri_after2 = rem_after2 * (rem_after2 - 1) / 2;
        int max_v = L - tri_after2;
        if (max_v <= u) continue;

        // Distances present after u:
        // {d_s, d_t, d_st, du0, du1, du2}
        for (int v = u + 1; v <= max_v; ++v) {
            int dv0 = v;       // v - 0
            int dv1 = v - s;   // v - s
            int dv2 = v - t;   // v - t
            int dv3 = v - u;   // v - u
            // compare to existing set
            if (dv0 == d_s || dv0 == d_t || dv0 == d_st || dv0 == du0 || dv0 == du1 || dv0 == du2) continue;
            if (dv1 == d_s || dv1 == d_t || dv1 == d_st || dv1 == du0 || dv1 == du1 || dv1 == du2) continue;
            if (dv2 == d_s || dv2 == d_t || dv2 == d_st || dv2 == du0 || dv2 == du1 || dv2 == du2) continue;
            if (dv3 == d_s || dv3 == d_t || dv3 == d_st || dv3 == du0 || dv3 == du1 || dv3 == du2) continue;
            // pairwise distinct among dv0..dv3; trivial order ensures dv0>dv1>dv2>dv3>0 but check collisions among dv* themselves:
            if (dv0 == dv1 || dv0 == dv2 || dv0 == dv3) continue;
            if (dv1 == dv2 || dv1 == dv3) continue;
            if (dv2 == dv3) continue;
            ok2 = 2; best_u = (best_u == 0 ? u : best_u); break;
        }
        if (ok2) break;
        if (!best_u) best_u = u; // remember first one-step-feasible u as fallback
    }
    ok[i] = ok2 ? ok2 : ok1;
    cands[i].u_hint = best_u;
}

/* ---------------- Heartbeat ---------------- */
static volatile int g_done = 0;
static volatile int g_current_L = -1;
static double g_vt_sec = 0.0;
static struct timespec g_ts_start;

static void *heartbeat_thread(void *)
{
    while (!g_done) {
        struct timespec ts_now; clock_gettime(CLOCK_MONOTONIC, &ts_now);
        double since = (ts_now.tv_sec - g_ts_start.tv_sec) + (ts_now.tv_nsec - g_ts_start.tv_nsec) / 1e9;
        int L = g_current_L;
        if (g_vt_sec > 0.0 && L >= 0) {
            // format mm:ss.mmm
            int minutes = (int)(since / 60.0);
            double seconds = since - minutes * 60.0;
            if (minutes > 0) fprintf(stderr, "[VT] %02d:%06.3f elapsed – current L=%d\n", minutes, seconds, L);
            else              fprintf(stderr, "[VT] %.3f s elapsed – current L=%d\n", seconds, L);
            fflush(stderr);
        }
        struct timespec req = { (time_t)g_vt_sec, (long)((g_vt_sec - (time_t)g_vt_sec) * 1e9) };
        nanosleep(&req, NULL);
    }
    return NULL;
}

/* ---------------- Optional async prefilter worker ---------------- */
struct PrefilterJob {
    int n;
    int L;
    long long total;
    Cand *cands; // input snapshot for device copy
    std::vector<unsigned char> ok_out;
    size_t ok2_cnt{0}, ok1_cnt{0};
    volatile int done{0};
    std::vector<Cand> cands_out; // device-updated copies (for u_hint)
};

static void *prefilter_worker(void *arg)
{
    PrefilterJob *job = (PrefilterJob*)arg;
    long long total = job->total;
    Cand *d_cands = nullptr; unsigned char *d_ok = nullptr; unsigned char *h_ok = nullptr;
    // bind device for this thread
    hipSetDevice(0);
    hipMalloc(&d_cands, sizeof(Cand) * (size_t)total);
    hipMalloc(&d_ok, (size_t)total);
    hipMemcpy(d_cands, job->cands, sizeof(Cand) * (size_t)total, hipMemcpyHostToDevice);
    int threads = 256; int blocks = (int)((total + threads - 1) / threads);
    prefilter_kernel<<<blocks, threads>>>(job->n, job->L, d_cands, total, d_ok);
    hipError_t kerr = hipGetLastError();
    if (kerr != hipSuccess) {
        fprintf(stderr, "[CUDA] prefilter kernel launch error: %s (%d)\n", hipGetErrorString(kerr), (int)kerr);
    }
    hipDeviceSynchronize();
    h_ok = (unsigned char*)malloc((size_t)total);
    hipMemcpy(h_ok, d_ok, (size_t)total, hipMemcpyDeviceToHost);
    // Copy device cands into job->cands_out to avoid racing writes to main cands
    job->cands_out.resize((size_t)total);
    hipMemcpy(job->cands_out.data(), d_cands, sizeof(Cand) * (size_t)total, hipMemcpyDeviceToHost);
    size_t ok2 = 0, ok1 = 0;
    for (size_t i = 0; i < (size_t)total; ++i) { if (h_ok[i] >= 2) ++ok2; else if (h_ok[i] == 1) ++ok1; }
    job->ok2_cnt = ok2; job->ok1_cnt = ok1;
    job->ok_out.assign(h_ok, h_ok + (size_t)total);
    hipFree(d_cands); hipFree(d_ok); free(h_ok);
    job->done = 1;
    return NULL;
}

/* ---------------- Main (CUDA-enhanced mp) ---------------- */
int main(int argc, char **argv)
{
    if (argc < 2) {
        fprintf(stderr, "Usage: %s <n> [-b] [-v] [-H] [-f <file>] [-fi <sec>] [-vt <min>] [-wu <N>] [-dh] [-dw <W>] [-ap]\n", argv[0]);
        return 1;
    }
    int n = atoi(argv[1]);
    bool verbose = false;
    bool use_b = false;
    bool hints = false; // enable LUT hint order and fast-lane only with -H
    const char *cp_path = NULL;
    int cp_interval = 60;
    double vt_min = 0.0;
    // Tunables: warmup size and depth-3 hinting
    long long warmup_limit = 8192; // default
    const char *env_wu = getenv("GOLOMB_WARMUP");
    if (env_wu && *env_wu) {
        long long v = atoll(env_wu); if (v > 0) warmup_limit = v;
    }
    int dfs3_hint = (getenv("GOLOMB_DFS3_HINT") != NULL) ? 1 : 0;
    int u_win = 16; const char *env_uw = getenv("GOLOMB_UWIN"); if (env_uw && *env_uw) { int v = atoi(env_uw); if (v > 0) u_win = v; }
    int async_pref = (getenv("GOLOMB_ASYNC_PREF") != NULL) ? 1 : 0;

    for (int i = 2; i < argc; ++i) {
        if (strcmp(argv[i], "-v") == 0) verbose = true;
        else if (strcmp(argv[i], "-b") == 0) use_b = true;
        else if (strcmp(argv[i], "-H") == 0) hints = true;
        else if (strcmp(argv[i], "-f") == 0 && i + 1 < argc) { cp_path = argv[++i]; }
        else if (strcmp(argv[i], "-fi") == 0 && i + 1 < argc) { cp_interval = atoi(argv[++i]); if (cp_interval <= 0) cp_interval = 60; }
        else if (strcmp(argv[i], "-vt") == 0 && i + 1 < argc) { vt_min = atof(argv[++i]); }
        else if (strcmp(argv[i], "-wu") == 0 && i + 1 < argc) { long long v = atoll(argv[++i]); if (v > 0) warmup_limit = v; }
        else if (strcmp(argv[i], "-dh") == 0) { dfs3_hint = 1; }
        else if (strcmp(argv[i], "-dw") == 0 && i + 1 < argc) { int v = atoi(argv[++i]); if (v > 0) u_win = v; }
        else if (strcmp(argv[i], "-ap") == 0) { async_pref = 1; }
        else {
            fprintf(stderr, "Unknown or incomplete option: %s\n", argv[i]);
            return 2;
        }
    }

    /* Enable SIMD in C dfs unless explicitly disabled. solver.c was built with -march=native. */
    g_use_simd = (getenv("GOLOMB_NO_SIMD") == NULL);

    const ruler_t *ref = lut_lookup_by_marks(n);
    if (!use_b || !ref) {
        fprintf(stderr, "This CUDA variant currently requires -b and a known LUT length for n=%d.\n", n);
        return 3;
    }
    int target_length = ref->length;

    // Start time and heartbeat
    clock_gettime(CLOCK_MONOTONIC, &g_ts_start);
    time_t t_start_wall = time(NULL);
    char start_iso[32]; strftime(start_iso, sizeof start_iso, "%F %T", localtime(&t_start_wall));
    fprintf(stderr, "Start time: %s\n", start_iso);
    g_current_L = target_length;
    g_vt_sec = vt_min > 0 ? vt_min * 60.0 : 0.0;
    pthread_t hb_th; if (g_vt_sec > 0.0) pthread_create(&hb_th, NULL, heartbeat_thread, NULL);

    // Build candidates like -mp
    int half = target_length / 2;
    int T = target_length - (n - 2);
    int second_max = half; if (second_max > T - 1) second_max = T - 1; if (second_max < 1) second_max = 1;

    long long total = 0;
    for (int s = 1; s <= second_max; ++s) {
        int cnt = T - s; if (cnt > 0) total += cnt;
    }
    std::vector<Cand> cands; cands.reserve((size_t)total);
    int use_hint_order = (hints && ref && getenv("GOLOMB_NO_HINTS") == NULL) ? 1 : 0;
    for (int s = 1; s <= second_max; ++s) {
        for (int t = s + 1; t <= T; ++t) {
            int score = 0;
            if (use_hint_order) {
                int ds = s - ref->pos[1]; if (ds < 0) ds = -ds;
                int dt = t - ref->pos[2]; if (dt < 0) dt = -dt;
                score = ds + dt;
            }
            cands.push_back({s, t, 0, score});
        }
    }
    if (use_hint_order && cands.size() > 1) {
        std::stable_sort(cands.begin(), cands.end(), [](const Cand &a, const Cand &b){
            if (a.score != b.score) return a.score < b.score;
            if (a.s != b.s) return a.s < b.s;
            return a.t < b.t;
        });
    }

    // Checkpoint bitset
    size_t words = (size_t)((total + 31) / 32); if (words == 0) words = 1;
    std::vector<uint32_t> done_words(words, 0);
    if (cp_path && *cp_path) {
        int hs = use_hint_order && ref ? ref->pos[1] : 0;
        int ht = use_hint_order && ref ? ref->pos[2] : 0;
        (void)cp_load_file(cp_path, n, target_length, total, hs, ht, use_hint_order, done_words.data(), words);
        (void)cp_save_file(cp_path, n, target_length, total, hs, ht, use_hint_order, done_words.data(), words);
    }

    // Optional GPU prefilter: mark candidates that can advance one or two steps without immediate duplicates
    std::vector<unsigned char> ok_host; ok_host.reserve((size_t)total);
    int device_count = 0;
    hipError_t derr = hipGetDeviceCount(&device_count);
    int rt_ver = 0, dr_ver = 0; hipRuntimeGetVersion(&rt_ver); hipDriverGetVersion(&dr_ver);
    const char *cvd = getenv("CUDA_VISIBLE_DEVICES");
    fprintf(stderr, "[CUDA] Runtime=%d Driver=%d CUDA_VISIBLE_DEVICES=%s\n", rt_ver, dr_ver, cvd ? cvd : "(unset)");
    if (derr != hipSuccess) {
        fprintf(stderr, "[CUDA] hipGetDeviceCount error: %s (%d)\n", hipGetErrorString(derr), (int)derr);
    }
    if (device_count > 0) {
        int dev = 0;
        hipError_t sderr = hipSetDevice(dev);
        if (sderr != hipSuccess) {
            fprintf(stderr, "[CUDA] hipSetDevice(%d) failed: %s (%d)\n", dev, hipGetErrorString(sderr), (int)sderr);
        }
        // create context
        hipFree(0);
        hipDeviceProp_t prop; hipGetDeviceProperties(&prop, dev);
        fprintf(stderr, "[CUDA] Using device %d: %s\n", dev, prop.name);
    } else {
        fprintf(stderr, "[CUDA] No CUDA device found – running CPU-only prefilter.\n");
    }
    // Enable GPU prefilter only when -H is NOT set; skip entirely under hints fast-lane
    pthread_t pf_th; PrefilterJob pf_job{}; int pf_started = 0;
    if (!use_hint_order && device_count > 0 && total > 0) {
        if (async_pref) {
            pf_job.n = n; pf_job.L = target_length; pf_job.total = total; pf_job.cands = cands.data();
            // Ensure CUDA context exists for the thread
            hipFree(0);
            pthread_create(&pf_th, NULL, prefilter_worker, &pf_job);
            pf_started = 1;
        } else {
            // synchronous prefilter
            Cand *d_cands = nullptr; unsigned char *d_ok = nullptr; unsigned char *h_ok = (unsigned char*)malloc((size_t)total);
            hipMalloc(&d_cands, sizeof(Cand) * (size_t)total);
            hipMalloc(&d_ok, (size_t)total);
            hipMemcpy(d_cands, cands.data(), sizeof(Cand) * (size_t)total, hipMemcpyHostToDevice);
            int threads = 256; int blocks = (int)((total + threads - 1) / threads);
            prefilter_kernel<<<blocks, threads>>>(n, target_length, d_cands, total, d_ok);
            hipError_t kerr = hipGetLastError();
            if (kerr != hipSuccess) {
                fprintf(stderr, "[CUDA] prefilter kernel launch error: %s (%d)\n", hipGetErrorString(kerr), (int)kerr);
            }
            hipDeviceSynchronize();
            hipMemcpy(h_ok, d_ok, (size_t)total, hipMemcpyDeviceToHost);
            hipMemcpy(cands.data(), d_cands, sizeof(Cand) * (size_t)total, hipMemcpyDeviceToHost);
            size_t ok2_cnt = 0, ok1_cnt = 0;
            for (size_t i = 0; i < (size_t)total; ++i) { if (h_ok[i] >= 2) ++ok2_cnt; else if (h_ok[i] == 1) ++ok1_cnt; }
            fprintf(stderr, "[CUDA] Prefiltered %lld candidates: %zu two-step, %zu one-step.\n", total, ok2_cnt, ok1_cnt);
            ok_host.assign(h_ok, h_ok + (size_t)total);
            hipFree(d_cands); hipFree(d_ok); free(h_ok);
        }
    }

    // Fast-lane: try exact LUT pair on CPU first only if hints are enabled (-H)
    volatile int found = 0; ruler_t res_local{};
    if (use_hint_order) {
        int s0 = ref->pos[1], t0 = ref->pos[2];
        if (s0 >= 1 && s0 <= second_max && t0 > s0 && t0 <= T) {
            uint64_t bs[BS_WORDS] = {0};
            int pos[MAX_MARKS]; pos[0] = 0; pos[1] = s0; pos[2] = t0;
            set_bit64(bs, s0);
            int d13 = t0; int d23 = t0 - s0;
            if (!test_bit64(bs, d13) && !test_bit64(bs, d23)) {
                set_bit64(bs, d13); set_bit64(bs, d23);
                if (dfs(3, n, target_length, pos, bs, verbose)) {
                    res_local.marks = n; res_local.length = pos[n - 1]; memcpy(res_local.pos, pos, n * sizeof(int)); found = 1;
                }
            }
        }
    }

    // Quick stage: try a larger prefix of candidates with dfs at depth=3 (pure DFS over (s,t)).
    // This leverages the GPU's feasibility flags early without global reordering.
    if (!found && total > 0) {
        long long warmup = total < warmup_limit ? total : warmup_limit;
        // Build warmup index list: when -H is not used, prioritize ok>=2 then ok==1 without reordering global cands
        std::vector<long long> warm_idx; warm_idx.reserve((size_t)warmup);
        if (!use_hint_order && ok_host.size() == (size_t)total) {
            for (long long i = 0; i < total && (long long)warm_idx.size() < warmup; ++i) if (ok_host[(size_t)i] >= 2) warm_idx.push_back(i);
            for (long long i = 0; i < total && (long long)warm_idx.size() < warmup; ++i) if (ok_host[(size_t)i] == 1) warm_idx.push_back(i);
            for (long long i = 0; i < total && (long long)warm_idx.size() < warmup; ++i) warm_idx.push_back(i);
        } else {
            for (long long i = 0; i < warmup; ++i) warm_idx.push_back(i);
        }
        #pragma omp parallel for schedule(dynamic, 16)
        for (long long j = 0; j < (long long)warm_idx.size(); ++j) {
            if (found) continue;
            long long i = warm_idx[(size_t)j];
            if (found) continue;
            int second = cands[(size_t)i].s;
            int third  = cands[(size_t)i].t;
            if (third - second == second) continue; // skip isosceles triangle
            uint64_t bs[BS_WORDS] = {0};
            int pos[MAX_MARKS]; pos[0] = 0; pos[1] = second; pos[2] = third;
            set_bit64(bs, second);
            int d13 = third; int d23 = third - second;
            if (test_bit64(bs, d13) || test_bit64(bs, d23)) continue;
            set_bit64(bs, d13); set_bit64(bs, d23);
            if (dfs(3, n, target_length, pos, bs, verbose)) {
                #pragma omp critical
                {
                    if (!found) { res_local.marks = n; res_local.length = pos[n - 1]; memcpy(res_local.pos, pos, n * sizeof(int)); found = 1; }
                }
            }
        }
    }

    // If prefilter was launched asynchronously, wait here so u_hint values are ready for the main loop
    if (pf_started) {
        pthread_join(pf_th, NULL);
        fprintf(stderr, "[CUDA] Prefiltered %lld candidates: %zu two-step, %zu one-step.\n", total, pf_job.ok2_cnt, pf_job.ok1_cnt);
        ok_host.swap(pf_job.ok_out);
        // merge u_hint back into main cands without changing order
        if (pf_job.cands_out.size() == (size_t)total) {
            for (size_t i = 0; i < (size_t)total; ++i) cands[i].u_hint = pf_job.cands_out[i].u_hint;
        }
    }

    struct timespec ts_last_flush; clock_gettime(CLOCK_MONOTONIC, &ts_last_flush);

    // Parallel CPU search across candidates (mp-like): pure DFS(3) over (s,t), respecting checkpoint
    #pragma omp parallel for schedule(dynamic, 16)
    for (long long i = 0; i < total; ++i) {
        if (found) continue;
        // skip processed
        if (cp_path && *cp_path) {
            size_t wi = (size_t)(i >> 5); uint32_t mask = 1u << (i & 31);
            if (done_words[wi] & mask) continue;
        }
        int second = cands[(size_t)i].s;
        int third  = cands[(size_t)i].t;
        if (third - second == second) goto checkpoint_update; // skip isosceles triangle
        // Initialize base state for depth=3. Optionally bias depth-3 by trying u near u_hint first, then call dfs(3)
        {
            uint64_t bs[BS_WORDS] = {0};
            set_bit64(bs, second);
            int d13 = third; int d23 = third - second;
            if (!test_bit64(bs, d13) && !test_bit64(bs, d23)) {
                set_bit64(bs, d13); set_bit64(bs, d23);
                int pos[MAX_MARKS]; pos[0] = 0; pos[1] = second; pos[2] = third;
                bool ok_found = false;
                if (dfs3_hint) {
                    // Compute bound for u (depth=4)
                    const int rem_after2 = n - 4;
                    const int tri_after2 = rem_after2 * (rem_after2 - 1) / 2;
                    const int max_u = target_length - tri_after2;
                    const int u_hint = cands[(size_t)i].u_hint;
                    auto try_u = [&](int u)->bool {
                        const int du0 = u;
                        const int du1 = u - second;
                        const int du2 = u - third;
                        if (du1 <= 0 || du2 <= 0) return false;
                        if (test_bit64(bs, du0) || test_bit64(bs, du1) || test_bit64(bs, du2)) return false;
                        if (du0 == du1 || du0 == du2 || du1 == du2) return false;
                        uint64_t bs_u[BS_WORDS]; memcpy(bs_u, bs, sizeof(bs_u));
                        set_bit64(bs_u, du0); set_bit64(bs_u, du1); set_bit64(bs_u, du2);
                        int pos_u[MAX_MARKS];
                        pos_u[0] = pos[0]; pos_u[1] = pos[1]; pos_u[2] = pos[2]; pos_u[3] = u;
                        if (dfs(4, n, target_length, pos_u, bs_u, verbose)) {
                            #pragma omp critical
                            {
                                if (!found) { res_local.marks = n; res_local.length = pos_u[n - 1]; memcpy(res_local.pos, pos_u, n * sizeof(int)); found = 1; }
                            }
                            return true;
                        }
                        return false;
                    };
                    if (!found && max_u > third && u_hint > third && u_hint <= max_u) {
                        if (try_u(u_hint)) ok_found = true;
                    }
                    if (!found && !ok_found && max_u > third && u_hint > 0) {
                        int start = std::max(third + 1, u_hint - u_win);
                        int end   = std::min(max_u, u_hint + u_win);
                        for (int u = start; u <= end && !found; ++u) {
                            if (u == u_hint) continue;
                            if (try_u(u)) { ok_found = true; break; }
                        }
                    }
                }
                if (!ok_found && dfs(3, n, target_length, pos, bs, verbose)) {
                    #pragma omp critical
                    {
                        if (!found) { res_local.marks = n; res_local.length = pos[n - 1]; memcpy(res_local.pos, pos, n * sizeof(int)); found = 1; }
                    }
                }
            }
        }
checkpoint_update:
        if (cp_path && *cp_path) {
            size_t wi = (size_t)(i >> 5); uint32_t mask = 1u << (i & 31);
            __sync_fetch_and_or(&done_words[wi], mask);
            struct timespec ts_now; clock_gettime(CLOCK_MONOTONIC, &ts_now);
            time_t dt = ts_now.tv_sec - ts_last_flush.tv_sec;
            if (dt >= cp_interval) {
                #pragma omp critical
                {
                    struct timespec ts_chk; clock_gettime(CLOCK_MONOTONIC, &ts_chk);
                    if (ts_chk.tv_sec - ts_last_flush.tv_sec >= cp_interval) {
                        int hs2 = use_hint_order && ref ? ref->pos[1] : 0;
                        int ht2 = use_hint_order && ref ? ref->pos[2] : 0;
                        (void)cp_save_file(cp_path, n, target_length, total, hs2, ht2, use_hint_order, done_words.data(), words);
                        ts_last_flush = ts_chk;
                    }
                }
            }
        }
    }

    if (cp_path && *cp_path) {
        int hs = use_hint_order && ref ? ref->pos[1] : 0;
        int ht = use_hint_order && ref ? ref->pos[2] : 0;
        (void)cp_save_file(cp_path, n, target_length, total, hs, ht, use_hint_order, done_words.data(), words);
    }

    g_done = 1; if (g_vt_sec > 0.0) pthread_join(hb_th, NULL);

    if (found) {
        // Print in the same format as the C variant:
        // length=..\nmarks=..\npositions=..\ndistances=..\nmissing=..
        int L = res_local.length;
        int m = res_local.marks;
        printf("length=%d\nmarks=%d\npositions=", L, m);
        for (int i = 0; i < m; ++i) {
            printf("%d%s", res_local.pos[i], (i == m - 1) ? "" : " ");
        }
        // Build distance presence 1..L
        std::vector<unsigned char> present((size_t)L + 1, 0);
        for (int j = 0; j < m; ++j) {
            for (int i = 0; i < j; ++i) {
                int d = res_local.pos[j] - res_local.pos[i];
                if (d >= 1 && d <= L) present[(size_t)d] = 1;
            }
        }
        // distances line
        printf("\ndistances=");
        for (int d = 1, first = 1; d <= L; ++d) {
            if (present[(size_t)d]) {
                if (!first) putchar(' ');
                printf("%d", d);
                first = 0;
            }
        }
        // missing line
        printf("\nmissing=");
        for (int d = 1, first = 1; d <= L; ++d) {
            if (!present[(size_t)d]) {
                if (!first) putchar(' ');
                printf("%d", d);
                first = 0;
            }
        }
        putchar('\n');
        return 0;
    }
    fprintf(stderr, "No ruler found at L=%d (unexpected for LUT-verified -b).\n", target_length);
    return 1;
}
